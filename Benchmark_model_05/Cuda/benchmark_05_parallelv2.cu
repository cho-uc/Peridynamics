#include "hip/hip_runtime.h"
/*
Serialized all kernels that are slower in GPU
	-cal_displacement,
	-pforce_reinitialization,
	-u_reinitialization 
 Serial result : 4882 ms (10240 nodes)
Parallel result : - ms
Parallel result with partial serialized: 3492 ms

*/
#include <cmath>	//for calculating power & NaN
#include<iostream>
#include<cstdio>
#include <vector>
#include <cstdlib>
#include <fstream> // for writing to file
#include <math.h>       //exp, pi
#include <chrono>	//for time measurement
#include <fstream>
#include <ctime>
#include "kernel_main.h"

using namespace std;
using namespace std::chrono;

int main(int argc, char **argv){
	cout<<"Start of program"<<endl;
	
	const float length = 1.0; //X
	const float width = 1.0e-1; //Y
	const float heigth = 1.0e-1; //Z
	const float load = 200.0e6; //Newton
	
	const size_t ndivx = 32*4;		//must be a multiply of 32
	//const size_t ndivx = 4;
	//const size_t ndivy = 3;
	const size_t ndivy = 10;
	//const size_t ndivz = 2;
	const size_t ndivz = 8;
	const size_t node = ndivx*ndivy*ndivz;
	
	size_t *neighbor_list_pointer= (size_t*) calloc (node, sizeof(size_t));
	size_t *neighbor_list= (size_t*) calloc (node*node/2, sizeof(size_t));//assume length node/2
	//size_t *neighbor_list= (size_t*) calloc (200*node, sizeof(size_t));//assume length 200
	
	const float delta=length/ndivx;
	const float small_delta=3.015*delta; //horizon
	const float delta_x=length/ndivx;
	const float delta_y=width/ndivy;
	const float delta_z=heigth/ndivz;
	
	float *x= (float*) calloc (node, sizeof(float));
	float *y= (float*) calloc (node, sizeof(float));
	float *z= (float*) calloc (node, sizeof(float));
	float *delta_V= (float*) calloc (node, sizeof(float));
	
	float *m= (float*) calloc (node, sizeof(float)); // weight
	
	float *x_plus_ux= (float*) calloc (node, sizeof(float));
	float *y_plus_uy= (float*) calloc (node, sizeof(float));
	float *z_plus_uz= (float*) calloc (node, sizeof(float));
	float *u_n1= (float*) calloc (node, sizeof(float));
	
	float *f_x= (float*) calloc (node, sizeof(float));
	float *f_y= (float*) calloc (node, sizeof(float));
	float *f_z= (float*) calloc (node, sizeof(float));
	
	float *b_x= (float*) calloc (node, sizeof(float));	//body force
	float *b_y= (float*) calloc (node, sizeof(float));
	float *b_z= (float*) calloc (node, sizeof(float));
	
	float *theta= (float*) calloc (node, sizeof(float)); 	//dilation
	
	float *u_x_n0= (float*) calloc (node, sizeof(float));
	float *u_y_n0= (float*) calloc (node, sizeof(float));
	float *u_z_n0= (float*) calloc (node, sizeof(float));
	
	float *u_x_n1= (float*) calloc (node, sizeof(float));
	float *u_y_n1= (float*) calloc (node, sizeof(float));
	float *u_z_n1= (float*) calloc (node, sizeof(float));
	
	float *u_dot_x_n0= (float*) calloc (node, sizeof(float));
	float *u_dot_y_n0= (float*) calloc (node, sizeof(float));
	float *u_dot_z_n0= (float*) calloc (node, sizeof(float));
	
	float *u_dot_x_nhalf = (float*) calloc (node, sizeof(float));
	float *u_dot_y_nhalf = (float*) calloc (node, sizeof(float));
	float *u_dot_z_nhalf = (float*) calloc (node, sizeof(float));
	
	float *u_doubledot_x_n0 = (float*) calloc (node, sizeof(float));
	float *u_doubledot_y_n0 = (float*) calloc (node, sizeof(float));
	float *u_doubledot_z_n0 = (float*) calloc (node, sizeof(float));
	
	const float E = 200.0e9; // Young's modulus
	const float nu=0.25; //Poisson's ratio
	const float mu=E/(2.0*(1.0+nu)); //shear modulus
	const float k_bulk_mod=E/(3.0*(1.0-2.0*nu)); // bulk modulus
	const float ro=7850.0; // mass densiy
	
	size_t iter_neighbor_list_pointer=0; 
	size_t iter_neighbor_list=0; //length of neighbor_list
	
	cout<<"No of nodes = "<<ndivx<<"x"<<ndivy<<"x"<<ndivz<<" = "<<node<<endl;

	
	for (size_t i = (node-4*(ndivy*ndivz)); i < node; ++i) {
		b_x[i] = load/delta_x; //load to end points
	}
	
	//########################################################################################
	//PARALLEL VERSION
	cout<<"Start parallel version"<<endl;
	
	int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    int gridSize;       // The actual grid size needed, based on input size 
	
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, discretize_blocks, 0, node); 
	gridSize = (node + blockSize - 1) / blockSize; 
	
	
	printf("\t discretize_blocks Blocksize= %i, ", blockSize);
	printf("minGridSize= %i, ",minGridSize);
	printf("gridSize= %i \n",gridSize);
	
	int blockSize_02; int minGridSize_02;  int gridSize_02;
	
	hipOccupancyMaxPotentialBlockSize(&minGridSize_02, &blockSize_02, weighted_vol, 0, node);
	gridSize_02 = (node + blockSize_02 - 1) / blockSize_02; 
	
	printf("\t weighted_vol= Blocksize= %i, ", blockSize_02);
	printf("minGridSize= %i, ",minGridSize_02);
	printf("gridSize= %i \n",gridSize_02);
	
	int blockSize_03; int minGridSize_03;  int gridSize_03;
	
	hipOccupancyMaxPotentialBlockSize(&minGridSize_03, &blockSize_03, cal_dilatation, 0, node);
	gridSize_03 = (node + blockSize_03 - 1) / blockSize_03;
	
	printf("\t cal_dilatation= Blocksize= %i, ", blockSize_03);
	printf("minGridSize= %i, ",minGridSize_03);
	printf("gridSize= %i \n",gridSize_03);
	
	dim3 gridDim_not_optimized(node/1024,1,1);         // 512 x 1 x 1
	dim3 blockDim_not_optimized(1024, 1,1); // 1024 x 1024 x 1

	
	printf ("Use gridDim = %i, ", gridDim_not_optimized.x);
	printf ("blockDim = %i \n", blockDim_not_optimized.x);
	
	hipDeviceSynchronize(); //CPU timer synchronization: synchronize CPU thread with GPU
	system_clock::time_point start_parallel = system_clock::now();
	// Create memory buffers on the device for each vector ------------------------
	
	size_t* buffer_neighbor_list_pointer;
	float* buffer_delta_V, * buffer_theta, * buffer_m;
	float* buffer_x, *buffer_y, *buffer_z;
	float* buffer_u_x_n0, * buffer_u_y_n0, * buffer_u_z_n0;
	float* buffer_u_x_n1,* buffer_u_y_n1,* buffer_u_z_n1;
	float* buffer_u_dot_x_n0, *buffer_u_dot_y_n0, *buffer_u_dot_z_n0;
	float* buffer_u_dot_x_nhalf,* buffer_u_dot_y_nhalf, * buffer_u_dot_z_nhalf;
	float* buffer_u_doubledot_x_n0,* buffer_u_doubledot_y_n0,* buffer_u_doubledot_z_n0;
	size_t* buffer_iter_neighbor_list, *buffer_node;
	float* buffer_small_delta, *buffer_delta_t;
	float* buffer_delta_x, * buffer_delta_y, * buffer_delta_z;
	size_t* buffer_ndivx, *buffer_ndivy, *buffer_ndivz;
	
	hipMalloc((void**)&buffer_neighbor_list_pointer, node*sizeof(size_t));
	hipMalloc((void**)&buffer_delta_V, node*sizeof(float));
	hipMalloc((void**)&buffer_theta, node*sizeof(float));
	hipMalloc((void**)&buffer_m, node*sizeof(float));
	hipMalloc((void**)&buffer_x, node*sizeof(float));
	hipMalloc((void**)&buffer_y, node*sizeof(float));
	hipMalloc((void**)&buffer_z, node*sizeof(float));
	hipMalloc((void**)&buffer_u_x_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_y_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_z_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_x_n1, node*sizeof(float));
	hipMalloc((void**)&buffer_u_y_n1, node*sizeof(float));
	hipMalloc((void**)&buffer_u_z_n1, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_x_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_y_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_z_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_x_nhalf, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_y_nhalf, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_z_nhalf, node*sizeof(float));
	hipMalloc((void**)&buffer_u_doubledot_x_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_doubledot_y_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_doubledot_z_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_iter_neighbor_list, sizeof(size_t));
	hipMalloc((void**)&buffer_node, sizeof(size_t));
	hipMalloc((void**)&buffer_small_delta, sizeof(float));
	hipMalloc((void**)&buffer_delta_x, sizeof(float));
	hipMalloc((void**)&buffer_delta_y, sizeof(float));
	hipMalloc((void**)&buffer_delta_z, sizeof(float));
	hipMalloc((void**)&buffer_delta_t, sizeof(float));
	hipMalloc((void**)&buffer_ndivx, sizeof(size_t));
	hipMalloc((void**)&buffer_ndivy, sizeof(size_t));
	hipMalloc((void**)&buffer_ndivz, sizeof(size_t));
	
	
	//buffer_neighbor_list is below as iter_neighbor_list is not yet known
			
	//Write buffer for initial values of the problem
	hipMemcpy(buffer_node, &node, sizeof(size_t), hipMemcpyHostToDevice);
	hipMemcpy(buffer_small_delta, &small_delta, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(buffer_delta_x, &delta_x, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(buffer_delta_y, &delta_y, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(buffer_delta_z, &delta_z, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(buffer_ndivx, &ndivx, sizeof(size_t), hipMemcpyHostToDevice);
	hipMemcpy(buffer_ndivy, &ndivy, sizeof(size_t), hipMemcpyHostToDevice);
	hipMemcpy(buffer_ndivz, &ndivz, sizeof(size_t), hipMemcpyHostToDevice);
	
	
	//Neighbor list search, critical delta_t, x, y, z, delta_V are below
	
	//Discretization
	
	/*discretize_blocks<<<gridSize, blockSize>>>(buffer_delta_x, buffer_delta_y, buffer_delta_z,
		buffer_x, buffer_y, buffer_z,
		buffer_delta_V, buffer_ndivx, buffer_ndivy, buffer_ndivz);
	*/
	
	discretize_blocks<<<gridDim_not_optimized, blockDim_not_optimized>>>(buffer_delta_x, buffer_delta_y, buffer_delta_z,
		buffer_x, buffer_y, buffer_z,
		buffer_delta_V, buffer_ndivx, buffer_ndivy, buffer_ndivz);
	
	hipMemcpy(x, buffer_x, node *sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(y, buffer_y, node *sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(z, buffer_z, node *sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(delta_V, buffer_delta_V, node *sizeof(float), hipMemcpyDeviceToHost);
	
	//Neighbor list search
	for(size_t i = 0; i < node; ++i){
		neighbor_list_pointer[i]=iter_neighbor_list_pointer;
		for(size_t j = 0; j < node; ++j){
			if (i!=j){
				float distance =sqrt(pow((x[i]-x[j]),2)+pow((y[i]-y[j]),2)+pow((z[i]-z[j]),2));
				if (distance<small_delta){
					neighbor_list[iter_neighbor_list] =j;
					iter_neighbor_list += 1;
					iter_neighbor_list_pointer +=1;
				}
			}
		}//end of j
	}
	cout<<"\t iter_neighbor_list= "<<iter_neighbor_list<<endl;
	
	size_t* buffer_neighbor_list;
	
	hipMemcpy(buffer_iter_neighbor_list, &iter_neighbor_list, sizeof(size_t), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&buffer_neighbor_list, iter_neighbor_list*sizeof(size_t));
	hipMemcpy(buffer_neighbor_list_pointer, neighbor_list_pointer, node*sizeof(size_t), hipMemcpyHostToDevice);
	hipMemcpy(buffer_neighbor_list, neighbor_list, iter_neighbor_list*sizeof(size_t), hipMemcpyHostToDevice);
	
	
	//Critical time step (improvement only 5 % in parallel version)
	
	float V_dot_C=0.0;
	float V_dot_C_temp=0.0;
	for (size_t i = 0; i < node; ++i) {	
			V_dot_C_temp=0.0; //Re-initialization
			
			size_t k_start=neighbor_list_pointer[i];
			size_t k_stop=0;
			if(i!=(node-1)){
				k_stop=neighbor_list_pointer[i+1];
			}
			if(i==(node-1)){
				k_stop=iter_neighbor_list;
			}
			for (size_t k = k_start; k < k_stop; ++k){
				size_t j=neighbor_list[k];
				float xi_x=x[j]-x[i];
				float xi_y=y[j]-y[i];
				float xi_z=z[j]-z[i];
				
				float xi_square=pow(xi_x,2.0)+pow(xi_y,2.0)+pow(xi_z,2.0);
				float C_p=18.0*k_bulk_mod/(sqrt(xi_square)*M_PI*pow(small_delta,4));
				V_dot_C_temp += (C_p*delta_V[j]);
			}
			if(V_dot_C_temp>V_dot_C){ //find max
				V_dot_C=V_dot_C_temp;
			}
	}
	
	//const float delta_t_critical=(length/ndivx)/sqrt(k/ro); //CLF method
	const float delta_t_critical=sqrt(2.0*ro/(V_dot_C));	
	
	cout << "delta_t_critical = "<<delta_t_critical<<endl;
	const float delta_t=delta_t_critical*1.0;	// safety factor = 1.0
	cout << "delta_t = "<<delta_t<<endl;
	
	hipMemcpy(buffer_delta_t, &delta_t, sizeof(float), hipMemcpyHostToDevice);
	
	//const float T=(100.0*delta_t);
	const float T=(200.0*delta_t);
	//const float T=(2.0*delta_t);
	const size_t num_steps= T/delta_t;
	
	//###########################################################################
	//Algo I (Linear Peridynamic Solid Initialization)	
	
	
	/*weighted_vol<<<gridSize_02, blockSize_02>>>(buffer_neighbor_list_pointer,
		buffer_neighbor_list, buffer_delta_V, buffer_m, 
		buffer_x, buffer_y, buffer_z,
		buffer_iter_neighbor_list,  buffer_node, buffer_small_delta);
	*/
	
	weighted_vol<<<gridDim_not_optimized, blockDim_not_optimized>>>( buffer_neighbor_list_pointer,
		buffer_neighbor_list, buffer_delta_V, buffer_m, 
		buffer_x, buffer_y, buffer_z,
		buffer_iter_neighbor_list,  buffer_node, buffer_small_delta);
	
	
	hipMemcpy(m, buffer_m, node *sizeof(float), hipMemcpyDeviceToHost);
	
	//Main kernel
	ofstream file_17;
	file_17.open ("disp_cpp.txt");
	
	for (size_t t_step = 0; t_step < num_steps; ++t_step){
		if(t_step%50==0){
			cout<<"Time step t=" <<t_step<< endl;
		}
		//First partial velocity update & nodal displacement (serialized due to slower in GPU)
		
		for (size_t i = 0; i < node; ++i) {
			u_dot_x_nhalf[i]=u_dot_x_n0[i]+(delta_t/2.0*u_doubledot_x_n0[i]);
			u_dot_y_nhalf[i]=u_dot_y_n0[i]+(delta_t/2.0*u_doubledot_y_n0[i]);
			u_dot_z_nhalf[i]=u_dot_z_n0[i]+(delta_t/2.0*u_doubledot_z_n0[i]);
			
			u_x_n1[i]=u_x_n0[i]+(delta_t*u_dot_x_nhalf[i]);
			u_y_n1[i]=u_y_n0[i]+(delta_t*u_dot_y_nhalf[i]);
			u_z_n1[i]=u_z_n0[i]+(delta_t*u_dot_z_nhalf[i]);
		}
		
		//Apply BC
		for (size_t j = 0; j <4*(ndivy*ndivz); ++j) { // in the beginning of the block
			u_x_n0[j]=0.0; 	u_y_n0[j]=0.0;	u_z_n0[j]=0.0;
			u_dot_x_n0[j]=0.0; 	u_dot_y_n0[j]=0.0;	u_dot_z_n0[j]=0.0;
			u_doubledot_x_n0[j]=0.0; u_doubledot_y_n0[j]=0.0; u_doubledot_z_n0[j]=0.0;
			u_dot_x_nhalf[j]=0.0; 	u_dot_y_nhalf[j]=0.0;	u_dot_z_nhalf[j]=0.0;
			u_x_n1[j]=0.0;          u_y_n1[j]=0.0;          u_z_n1[j]=0.0;
		}
		
		//Compute the dilatation using u at (n+1)
		hipMemcpy(buffer_u_x_n1, u_x_n1, sizeof(float) * node, hipMemcpyHostToDevice);
		hipMemcpy(buffer_u_y_n1, u_y_n1, sizeof(float) * node, hipMemcpyHostToDevice);
		hipMemcpy(buffer_u_z_n1, u_z_n1, sizeof(float) * node, hipMemcpyHostToDevice);
		
		
		/*cal_dilatation<<<gridSize_03, blockSize_03>>>( buffer_neighbor_list_pointer,
			buffer_neighbor_list, buffer_delta_V, 
			buffer_theta, buffer_m,
			buffer_x, buffer_y, buffer_z,
			buffer_u_x_n1, buffer_u_y_n1, buffer_u_z_n1,
			buffer_iter_neighbor_list, buffer_node,
			buffer_small_delta);*/
			
		cal_dilatation<<<gridDim_not_optimized, blockDim_not_optimized>>>( buffer_neighbor_list_pointer,
			buffer_neighbor_list, buffer_delta_V, 
			buffer_theta, buffer_m,
			buffer_x, buffer_y, buffer_z,
			buffer_u_x_n1, buffer_u_y_n1, buffer_u_z_n1,
			buffer_iter_neighbor_list, buffer_node,
			buffer_small_delta);
			
		hipMemcpy(theta, buffer_theta, node *sizeof(float), hipMemcpyDeviceToHost);	
	
		//Re-initialization peridynamics force (serialized due to slower in GPU)
		for (size_t i = 0; i < node; ++i) {
			f_x[i]=0.0; 	f_y[i]=0.0; 	f_z[i]=0.0;
		}
		
		//Compute the pairwise contributions to the global force density vector
		for (size_t i = 0; i < node; ++i) {
			size_t k_start=neighbor_list_pointer[i];
			size_t k_stop=0;
			if(i!=(node-1)){
				k_stop=neighbor_list_pointer[i+1];
			}
			if(i==(node-1)){
				k_stop=iter_neighbor_list;
			}
			for (size_t k = k_start; k < k_stop; ++k){
				size_t j=neighbor_list[k];
				
				float xi_x=x[j]-x[i];
				float xi_y=y[j]-y[i];
				float xi_z=z[j]-z[i];
				
				float eta_x=u_x_n1[j]-u_x_n1[i];
				float eta_y=u_y_n1[j]-u_y_n1[i];
				float eta_z=u_z_n1[j]-u_z_n1[i];
				
				float xi_square=pow(xi_x,2)+pow(xi_y,2)+pow(xi_z,2);
				float omega=exp(-xi_square/(small_delta*small_delta));
				float xi_plus_eta=sqrt(pow((xi_x+eta_x),2)+pow((xi_y+eta_y),2)+pow((xi_z+eta_z),2));
				float e=xi_plus_eta-sqrt(xi_square);	//extension state			
				
				
				float e_d=e-(theta[i]*sqrt(xi_square)/3.0);	//deviatoric extension state
				float t=(3.0/m[i]*k_bulk_mod*theta[i]*omega*sqrt(xi_square))+(15.0*mu/m[i]*omega*e_d);
				float M_x=(xi_x+eta_x)/xi_plus_eta;
				float M_y=(xi_y+eta_y)/xi_plus_eta;
				float M_z=(xi_z+eta_z)/xi_plus_eta;
				
				f_x[i] = f_x[i]+(t*M_x*delta_V[j]);
				f_y[i] = f_y[i]+(t*M_y*delta_V[j]);
				f_z[i] = f_z[i]+(t*M_z*delta_V[j]);
				
				f_x[j] = f_x[j]-(t*M_x*delta_V[i]);
				f_y[j] = f_y[j]-(t*M_y*delta_V[i]);
				f_z[j] = f_z[j]-(t*M_z*delta_V[i]);
			}
			
		}
		
		//Calculate displacement (serialized due to slower in GPU)
		for (size_t i = 0; i < node; ++i) {
			float u_doubledot_x_n1=(f_x[i]+b_x[i])/ro; //no need to use array
			float u_doubledot_y_n1=(f_y[i]+b_y[i])/ro;
			float u_doubledot_z_n1=(f_z[i]+b_z[i])/ro;
			
			
			float u_dot_x_n1=u_dot_x_nhalf[i]+(delta_t/2.0*u_doubledot_x_n1); //no need to use array
			float u_dot_y_n1=u_dot_y_nhalf[i]+(delta_t/2.0*u_doubledot_y_n1);
			float u_dot_z_n1=u_dot_z_nhalf[i]+(delta_t/2.0*u_doubledot_z_n1);
			
			
			//Re-initialization
			
			u_x_n0[i]=u_x_n1[i];
			u_y_n0[i]=u_y_n1[i];
			u_z_n0[i]=u_z_n1[i];
			u_dot_x_n0[i]=u_dot_x_n1;
			u_dot_y_n0[i]=u_dot_y_n1;
			u_dot_z_n0[i]=u_dot_z_n1;
			u_doubledot_x_n0[i]=u_doubledot_x_n1; 
			u_doubledot_y_n0[i]=u_doubledot_y_n1;
			u_doubledot_z_n0[i]=u_doubledot_z_n1;
			
		}
	
		
		file_17 <<t_step<<"   "<<u_x_n1[(node/2)]<<"   "<<u_x_n1[node-1]<<endl; //disp at end of rope
		
	} //end of time integration
	file_17.close();
	
	hipDeviceSynchronize(); //CPU timer synchronization: synchronize CPU thread with GPU
	system_clock::time_point stop_parallel = system_clock::now();
	std::chrono::duration<float, std::milli> duration_parallel = stop_parallel - start_parallel;
	cout << "Parallel peridynamics = "<<duration_parallel.count()<<" millisecond"<<endl;
	
	
	for (size_t i = 0; i < node; ++i) {
		u_n1[i]=sqrt(pow(u_x_n1[i],2.0)+pow(u_y_n1[i],2.0)+pow(u_z_n1[i],2.0));
		x_plus_ux[i]=x[i]+u_x_n1[i];
		y_plus_uy[i]=y[i]+u_y_n1[i];
		z_plus_uz[i]=z[i]+u_z_n1[i];
	}
	
	float u_n1_sum=0.0;
	for (size_t i = 0; i < node; ++i) {
		u_n1_sum += u_n1[i];
	}
	cout<<"u_n1_sum at the end of time step = "<< u_n1_sum<<endl;
	
	
	ofstream file_18;
	file_18.open ("pos_vs_disp.txt");
	for (size_t i = 0; i < node; ++i) {
		file_18 <<x[i]<<"   "<<u_x_n1[i]<<"   "<<u_y_n1[i]<<"   "<<u_z_n1[i]<<"   "<<u_n1[i];
		if(i < (node - 1)) {
			file_18 <<endl;
		}
		
	}
	file_18.close();
	
	hipFree(buffer_neighbor_list);
	hipFree(buffer_neighbor_list_pointer);
	hipFree(buffer_delta_V); hipFree(buffer_theta); hipFree(buffer_m);
	hipFree(buffer_x); hipFree(buffer_y); hipFree(buffer_z);
	hipFree(buffer_u_x_n0); hipFree(buffer_u_y_n0); hipFree(buffer_u_z_n0);
	hipFree(buffer_u_x_n1); hipFree(buffer_u_y_n1); hipFree(buffer_u_z_n1);
	hipFree(buffer_u_dot_x_n0); hipFree(buffer_u_dot_y_n0); hipFree(buffer_u_dot_z_n0);
	hipFree(buffer_u_dot_x_nhalf); hipFree(buffer_u_dot_y_nhalf); hipFree(buffer_u_dot_z_nhalf);
	hipFree(buffer_u_doubledot_x_n0); hipFree(buffer_u_doubledot_y_n0); hipFree(buffer_u_doubledot_z_n0);
	hipFree(buffer_iter_neighbor_list); hipFree(buffer_node); 
	hipFree(buffer_small_delta); hipFree(buffer_delta_t);
	hipFree(buffer_delta_x); hipFree(buffer_delta_y);  hipFree(buffer_delta_z);
	hipFree(buffer_ndivx); hipFree(buffer_ndivy);  hipFree(buffer_ndivz);
	
	
	free(neighbor_list);
	free(neighbor_list_pointer);
	free(f_x); free(f_y); free(f_z);
	free(b_x); free(b_y); free(b_z);
	free(x); free(y); free(z);
	free(delta_V);
	free(m);
	free(x_plus_ux); free(y_plus_uy);free(z_plus_uz);
	free(u_n1);
	free(theta);
	free(u_x_n0); free(u_y_n0); free(u_z_n0);
	free(u_x_n1); free(u_y_n1); free(u_z_n1);
	free(u_dot_x_n0); free(u_dot_y_n0); free(u_dot_z_n0);
	free(u_dot_x_nhalf); free(u_dot_y_nhalf); free(u_dot_z_nhalf);
	free(u_doubledot_x_n0); free(u_doubledot_y_n0); free(u_doubledot_z_n0);
	
	printf("End of program!");
	
	
}