#include "hip/hip_runtime.h"
/*
Serialized all kernels that are slower in GPU
	-cal_displacement,
	-pforce_reinitialization,
	-u_reinitialization 
Use Pinned-Memory in Host

Serial result : 4882 ms (10240 nodes)
Parallel result : - ms
Parallel result with partial serialized: 3492 ms

*/
#include <cmath>	//for calculating power & NaN
#include<iostream>
#include<cstdio>
#include <vector>
#include <cstdlib>
#include <fstream> // for writing to file
#include <math.h>       //exp, pi
#include <chrono>	//for time measurement
#include <fstream>
#include <ctime>
#include "kernel_main.h"

using namespace std;
using namespace std::chrono;

int main(int argc, char **argv){
	cout<<"Start of program"<<endl;
	
	const float length = 1.0; //X
	const float width = 1.0e-1; //Y
	const float heigth = 1.0e-1; //Z
	const float load = 200.0e6; //Newton
	
	const size_t ndivx = 32*4;		//must be a multiply of 32
	//const size_t ndivx = 4;
	//const size_t ndivy = 3;
	const size_t ndivy = 10;
	//const size_t ndivz = 2;
	const size_t ndivz = 8;
	const size_t node = ndivx*ndivy*ndivz;
	
	size_t *neighbor_list_pointer = new size_t[node];
	size_t *neighbor_list = new size_t[node*node/2]; //assume length node/2
	//size_t *neighbor_list = new size_t[node*300];//assume length 200
	
	const float delta=length/ndivx;
	const float small_delta=3.015*delta; //horizon
	const float delta_x=length/ndivx;
	const float delta_y=width/ndivy;
	const float delta_z=heigth/ndivz;
	
	float *x ;	float *y ; float *z ;
	float *delta_V;
	
	hipHostMalloc((void**)&x, node*sizeof(float));
	hipHostMalloc((void**)&y, node*sizeof(float));
	hipHostMalloc((void**)&z, node*sizeof(float));
	hipHostMalloc((void**)&delta_V, node*sizeof(float));
	
	float *m ; // weight
	
	hipHostMalloc((void**)&m, node*sizeof(float));
	
	float *x_plus_ux = new float[node];
	float *y_plus_uy = new float[node];
	float *z_plus_uz = new float[node];
	float *u_n1 = new float[node];
	
	float *f_x;	float *f_y;	float *f_z;
	
	hipHostMalloc((void**)&f_x, node*sizeof(float));
	hipHostMalloc((void**)&f_y, node*sizeof(float));
	hipHostMalloc((void**)&f_z, node*sizeof(float));
	
	float *b_x;	float *b_y;	float *b_z; //body force
	
	hipHostMalloc((void**)&b_x, node*sizeof(float));
	hipHostMalloc((void**)&b_y, node*sizeof(float));
	hipHostMalloc((void**)&b_z, node*sizeof(float));
	
	float *theta;	//dilation	
	
	hipHostMalloc((void**)&theta, node*sizeof(float));
	
	float *u_x_n0 ; float *u_x_n1 ;
	float *u_y_n0 ;	float *u_y_n1 ;
	float *u_z_n0 ; float *u_z_n1 ;	
	
	hipHostMalloc((void**)&u_x_n0, node*sizeof(float));
	hipHostMalloc((void**)&u_y_n0, node*sizeof(float));
	hipHostMalloc((void**)&u_z_n0, node*sizeof(float));
	hipHostMalloc((void**)&u_x_n1, node*sizeof(float));
	hipHostMalloc((void**)&u_y_n1, node*sizeof(float));
	hipHostMalloc((void**)&u_z_n1, node*sizeof(float));
	
	
	float *u_dot_x_n0;	float *u_dot_x_nhalf;
	float *u_dot_y_n0;	float *u_dot_y_nhalf;
	float *u_dot_z_n0;	float *u_dot_z_nhalf;
	
	hipHostMalloc((void**)&u_dot_x_n0, node*sizeof(float));
	hipHostMalloc((void**)&u_dot_y_n0, node*sizeof(float));
	hipHostMalloc((void**)&u_dot_z_n0, node*sizeof(float));
	hipHostMalloc((void**)&u_dot_x_nhalf, node*sizeof(float));
	hipHostMalloc((void**)&u_dot_y_nhalf, node*sizeof(float));
	hipHostMalloc((void**)&u_dot_z_nhalf, node*sizeof(float));
	
	float *u_doubledot_x_n0; 
	float *u_doubledot_y_n0; 
	float *u_doubledot_z_n0; 
	
	hipHostMalloc((void**)&u_doubledot_x_n0, node*sizeof(float));
	hipHostMalloc((void**)&u_doubledot_y_n0, node*sizeof(float));
	hipHostMalloc((void**)&u_doubledot_z_n0, node*sizeof(float));
	
	//------------------------------------------------
	
	hipMemset(b_x, 0, node*sizeof(float));
	hipMemset(b_y, 0, node*sizeof(float));
	hipMemset(b_z, 0, node*sizeof(float));
	
	/*hipMemset(f_x, 0, node*sizeof(float));
	hipMemset(f_y, 0, node*sizeof(float));
	hipMemset(f_z, 0, node*sizeof(float));
	
	hipMemset(theta, 0, node*sizeof(float));
	
	hipMemset(u_x_n0, 0, node*sizeof(float));
	hipMemset(u_y_n0, 0, node*sizeof(float));
	hipMemset(u_z_n0, 0, node*sizeof(float));
	hipMemset(u_x_n1, 0, node*sizeof(float));
	hipMemset(u_y_n1, 0, node*sizeof(float));
	hipMemset(u_z_n1, 0, node*sizeof(float));
	
	hipMemset(u_dot_x_n0, 0, node*sizeof(float));
	hipMemset(u_dot_y_n0, 0, node*sizeof(float));
	hipMemset(u_dot_z_n0, 0, node*sizeof(float));
	hipMemset(u_dot_x_nhalf, 0, node*sizeof(float));
	hipMemset(u_dot_y_nhalf, 0, node*sizeof(float));
	hipMemset(u_dot_z_nhalf, 0, node*sizeof(float));
	
	hipMemset(u_doubledot_x_n0, 0, node*sizeof(float));
	hipMemset(u_doubledot_y_n0, 0, node*sizeof(float));
	hipMemset(u_doubledot_z_n0, 0, node*sizeof(float));*/
	
	
	const float E = 200.0e9; // Young's modulus
	const float nu=0.25; //Poisson's ratio
	const float mu=E/(2.0*(1.0+nu)); //shear modulus
	const float k_bulk_mod=E/(3.0*(1.0-2.0*nu)); // bulk modulus
	const float ro=7850.0; // mass densiy
	
	size_t iter_neighbor_list_pointer=0; 
	size_t iter_neighbor_list=0; //length of neighbor_list
	
	cout<<"No of nodes = "<<ndivx<<"x"<<ndivy<<"x"<<ndivz<<" = "<<node<<endl;
	
	for (size_t i = (node-4*(ndivy*ndivz)); i < node; ++i) {
		b_x[i] = load/delta_x; //load to end points
	}
	
	//########################################################################################
	//PARALLEL VERSION
	cout<<"Start parallel version with pinned memory"<<endl;
	
	int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    int gridSize;       // The actual grid size needed, based on input size 
	
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, discretize_blocks, 0, node); 
	gridSize = (node + blockSize - 1) / blockSize; 
	
	
	printf("\t discretize_blocks Blocksize= %i, ", blockSize);
	printf("minGridSize= %i, ",minGridSize);
	printf("gridSize= %i \n",gridSize);
	
	int blockSize_02; int minGridSize_02;  int gridSize_02;
	
	hipOccupancyMaxPotentialBlockSize(&minGridSize_02, &blockSize_02, weighted_vol, 0, node);
	gridSize_02 = (node + blockSize_02 - 1) / blockSize_02; 
	
	printf("\t weighted_vol= Blocksize= %i, ", blockSize_02);
	printf("minGridSize= %i, ",minGridSize_02);
	printf("gridSize= %i \n",gridSize_02);
	
	int blockSize_03; int minGridSize_03;  int gridSize_03;
	
	hipOccupancyMaxPotentialBlockSize(&minGridSize_03, &blockSize_03, cal_dilatation, 0, node);
	gridSize_03 = (node + blockSize_03 - 1) / blockSize_03;
	
	printf("\t cal_dilatation= Blocksize= %i, ", blockSize_03);
	printf("minGridSize= %i, ",minGridSize_03);
	printf("gridSize= %i \n",gridSize_03);
	
	dim3 gridDim_not_optimized(node/1024,1,1);         // 512 x 1 x 1
	dim3 blockDim_not_optimized(1024, 1,1); // 1024 x 1024 x 1
	
	printf ("Use gridDim = %i, ", gridDim_not_optimized.x);
	printf ("blockDim = %i \n", blockDim_not_optimized.x);
	
	hipDeviceSynchronize(); //CPU timer synchronization: synchronize CPU thread with GPU
	system_clock::time_point start_parallel = system_clock::now();
	// Create memory buffers on the device for each vector ------------------------
	
	size_t* buffer_neighbor_list_pointer;
	float* buffer_delta_V, * buffer_theta, * buffer_m;
	float* buffer_x, *buffer_y, *buffer_z;
	float* buffer_u_x_n0, * buffer_u_y_n0, * buffer_u_z_n0;
	float* buffer_u_x_n1,* buffer_u_y_n1,* buffer_u_z_n1;
	float* buffer_u_dot_x_n0, *buffer_u_dot_y_n0, *buffer_u_dot_z_n0;
	float* buffer_u_dot_x_nhalf,* buffer_u_dot_y_nhalf, * buffer_u_dot_z_nhalf;
	float* buffer_u_doubledot_x_n0,* buffer_u_doubledot_y_n0,* buffer_u_doubledot_z_n0;
	size_t* buffer_iter_neighbor_list, *buffer_node;
	float* buffer_small_delta, *buffer_delta_t;
	float* buffer_delta_x, * buffer_delta_y, * buffer_delta_z;
	size_t* buffer_ndivx, *buffer_ndivy, *buffer_ndivz;
	
	hipMalloc((void**)&buffer_neighbor_list_pointer, node*sizeof(size_t));
	hipMalloc((void**)&buffer_delta_V, node*sizeof(float));
	hipMalloc((void**)&buffer_theta, node*sizeof(float));
	hipMalloc((void**)&buffer_m, node*sizeof(float));
	hipMalloc((void**)&buffer_x, node*sizeof(float));
	hipMalloc((void**)&buffer_y, node*sizeof(float));
	hipMalloc((void**)&buffer_z, node*sizeof(float));
	hipMalloc((void**)&buffer_u_x_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_y_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_z_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_x_n1, node*sizeof(float));
	hipMalloc((void**)&buffer_u_y_n1, node*sizeof(float));
	hipMalloc((void**)&buffer_u_z_n1, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_x_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_y_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_z_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_x_nhalf, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_y_nhalf, node*sizeof(float));
	hipMalloc((void**)&buffer_u_dot_z_nhalf, node*sizeof(float));
	hipMalloc((void**)&buffer_u_doubledot_x_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_doubledot_y_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_u_doubledot_z_n0, node*sizeof(float));
	hipMalloc((void**)&buffer_iter_neighbor_list, sizeof(size_t));
	hipMalloc((void**)&buffer_node, sizeof(size_t));
	hipMalloc((void**)&buffer_small_delta, sizeof(float));
	hipMalloc((void**)&buffer_delta_x, sizeof(float));
	hipMalloc((void**)&buffer_delta_y, sizeof(float));
	hipMalloc((void**)&buffer_delta_z, sizeof(float));
	hipMalloc((void**)&buffer_delta_t, sizeof(float));
	hipMalloc((void**)&buffer_ndivx, sizeof(size_t));
	hipMalloc((void**)&buffer_ndivy, sizeof(size_t));
	hipMalloc((void**)&buffer_ndivz, sizeof(size_t));
	
	
	//buffer_neighbor_list is below as iter_neighbor_list is not yet known
			
	//Write buffer for initial values of the problem
	hipMemcpy(buffer_node, &node, sizeof(size_t), hipMemcpyHostToDevice);
	hipMemcpy(buffer_small_delta, &small_delta, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(buffer_delta_x, &delta_x, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(buffer_delta_y, &delta_y, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(buffer_delta_z, &delta_z, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(buffer_ndivx, &ndivx, sizeof(size_t), hipMemcpyHostToDevice);
	hipMemcpy(buffer_ndivy, &ndivy, sizeof(size_t), hipMemcpyHostToDevice);
	hipMemcpy(buffer_ndivz, &ndivz, sizeof(size_t), hipMemcpyHostToDevice);
	
	
	//Neighbor list search, critical delta_t, x, y, z, delta_V are below
	
	//Discretization

	
	discretize_blocks<<<gridDim_not_optimized, blockDim_not_optimized>>>(buffer_delta_x, buffer_delta_y, buffer_delta_z,
		buffer_x, buffer_y, buffer_z,
		buffer_delta_V, buffer_ndivx, buffer_ndivy, buffer_ndivz);
		
	hipMemcpy(x, buffer_x, node *sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(y, buffer_y, node *sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(z, buffer_z, node *sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(delta_V, buffer_delta_V, node *sizeof(float), hipMemcpyDeviceToHost);
	
	//Neighbor list search
	for(size_t i = 0; i < node; ++i){
		neighbor_list_pointer[i]=iter_neighbor_list_pointer;
		for(size_t j = 0; j < node; ++j){
			if (i!=j){
				float distance =sqrt(pow((x[i]-x[j]),2)+pow((y[i]-y[j]),2)+pow((z[i]-z[j]),2));
				if (distance<small_delta){
					neighbor_list[iter_neighbor_list] =j;
					iter_neighbor_list += 1;
					iter_neighbor_list_pointer +=1;
				}
			}
		}//end of j
	}
	cout<<"\t iter_neighbor_list= "<<iter_neighbor_list<<endl;
	
	size_t* buffer_neighbor_list;
	
	hipMemcpy(buffer_iter_neighbor_list, &iter_neighbor_list, sizeof(size_t), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&buffer_neighbor_list, iter_neighbor_list*sizeof(size_t));

	hipMemcpy(buffer_neighbor_list_pointer, neighbor_list_pointer, node*sizeof(size_t), hipMemcpyHostToDevice);
	hipMemcpy(buffer_neighbor_list, neighbor_list, iter_neighbor_list*sizeof(size_t), hipMemcpyHostToDevice);
	//hipDeviceSynchronize();
	
	//Critical time step (improvement only 5 % in parallel version)
	
	
	float V_dot_C=0.0;
	float V_dot_C_temp=0.0;
	for (size_t i = 0; i < node; ++i) {	
			V_dot_C_temp=0.0; //Re-initialization
			
			size_t k_start=neighbor_list_pointer[i];
			size_t k_stop=0;
			if(i!=(node-1)){
				k_stop=neighbor_list_pointer[i+1];
			}
			if(i==(node-1)){
				k_stop=iter_neighbor_list;
			}
			for (size_t k = k_start; k < k_stop; ++k){
				size_t j=neighbor_list[k];
				float xi_x=x[j]-x[i];
				float xi_y=y[j]-y[i];
				float xi_z=z[j]-z[i];
				
				float xi_square=pow(xi_x,2.0)+pow(xi_y,2.0)+pow(xi_z,2.0);
				float C_p=18.0*k_bulk_mod/(sqrt(xi_square)*M_PI*pow(small_delta,4));
				V_dot_C_temp += (C_p*delta_V[j]);
			}
			if(V_dot_C_temp>V_dot_C){ //find max
				V_dot_C=V_dot_C_temp;
			}
	}
	
	//const float delta_t_critical=(length/ndivx)/sqrt(k/ro); //CLF method
	const float delta_t_critical=sqrt(2.0*ro/(V_dot_C));	
	
	cout << "delta_t_critical = "<<delta_t_critical<<endl;
	const float delta_t=delta_t_critical*1.0;	// safety factor = 1.0
	cout << "delta_t = "<<delta_t<<endl;
	
	hipMemcpy(buffer_delta_t, &delta_t, sizeof(float), hipMemcpyHostToDevice);
	
	//const float T=(100.0*delta_t);
	const float T=(200.0*delta_t);
	//const float T=(2.0*delta_t);
	const size_t num_steps= T/delta_t;
	
	//###########################################################################
	//Algo I (Linear Peridynamic Solid Initialization)	
	
	weighted_vol<<<gridDim_not_optimized, blockDim_not_optimized>>>( buffer_neighbor_list_pointer,
		buffer_neighbor_list, buffer_delta_V, buffer_m, 
		buffer_x, buffer_y, buffer_z,
		buffer_iter_neighbor_list,  buffer_node, buffer_small_delta);
	
	
	hipMemcpy(m, buffer_m, node *sizeof(float), hipMemcpyDeviceToHost);
	
	//Main kernel
	ofstream file_17;
	file_17.open ("disp_cpp.txt");
	
	for (size_t t_step = 0; t_step < num_steps; ++t_step){
		if(t_step%50==0){
			cout<<"Time step t=" <<t_step<< endl;
		}
		//First partial velocity update & nodal displacement (serialized due to slower in GPU)
		
		for (size_t i = 0; i < node; ++i) {
			u_dot_x_nhalf[i]=u_dot_x_n0[i]+(delta_t/2.0*u_doubledot_x_n0[i]);
			u_dot_y_nhalf[i]=u_dot_y_n0[i]+(delta_t/2.0*u_doubledot_y_n0[i]);
			u_dot_z_nhalf[i]=u_dot_z_n0[i]+(delta_t/2.0*u_doubledot_z_n0[i]);
			
			u_x_n1[i]=u_x_n0[i]+(delta_t*u_dot_x_nhalf[i]);
			u_y_n1[i]=u_y_n0[i]+(delta_t*u_dot_y_nhalf[i]);
			u_z_n1[i]=u_z_n0[i]+(delta_t*u_dot_z_nhalf[i]);
		}
		
	
	
		//Apply BC
		for (size_t j = 0; j <4*(ndivy*ndivz); ++j) { // in the beginning of the block
			u_x_n0[j]=0.0; 	u_y_n0[j]=0.0;	u_z_n0[j]=0.0;
			u_dot_x_n0[j]=0.0; 	u_dot_y_n0[j]=0.0;	u_dot_z_n0[j]=0.0;
			u_doubledot_x_n0[j]=0.0; u_doubledot_y_n0[j]=0.0; u_doubledot_z_n0[j]=0.0;
			u_dot_x_nhalf[j]=0.0; 	u_dot_y_nhalf[j]=0.0;	u_dot_z_nhalf[j]=0.0;
			u_x_n1[j]=0.0;          u_y_n1[j]=0.0;          u_z_n1[j]=0.0;
		}
			
		//Compute the dilatation using u at (n+1)
		hipMemcpy(buffer_u_x_n1, u_x_n1, sizeof(float) * node, hipMemcpyHostToDevice);
		hipMemcpy(buffer_u_y_n1, u_y_n1, sizeof(float) * node, hipMemcpyHostToDevice);
		hipMemcpy(buffer_u_z_n1, u_z_n1, sizeof(float) * node, hipMemcpyHostToDevice);
			
		cal_dilatation<<<gridDim_not_optimized, blockDim_not_optimized>>>( buffer_neighbor_list_pointer,
			buffer_neighbor_list, buffer_delta_V, 
			buffer_theta, buffer_m,
			buffer_x, buffer_y, buffer_z,
			buffer_u_x_n1, buffer_u_y_n1, buffer_u_z_n1,
			buffer_iter_neighbor_list, buffer_node,
			buffer_small_delta);
		hipMemcpy(theta, buffer_theta, node *sizeof(float), hipMemcpyDeviceToHost);
		
		//Re-initialization peridynamics force (serialized due to slower in GPU)
		for (size_t i = 0; i < node; ++i) {
			f_x[i]=0.0; 	f_y[i]=0.0; 	f_z[i]=0.0;
		}
		
		//Compute the pairwise contributions to the global force density vector
		for (size_t i = 0; i < node; ++i) {
			size_t k_start=neighbor_list_pointer[i];
			size_t k_stop=0;
			if(i!=(node-1)){
				k_stop=neighbor_list_pointer[i+1];
			}
			if(i==(node-1)){
				k_stop=iter_neighbor_list;
			}
			for (size_t k = k_start; k < k_stop; ++k){
				size_t j=neighbor_list[k];
				
				float xi_x=x[j]-x[i];
				float xi_y=y[j]-y[i];
				float xi_z=z[j]-z[i];
				
				float eta_x=u_x_n1[j]-u_x_n1[i];
				float eta_y=u_y_n1[j]-u_y_n1[i];
				float eta_z=u_z_n1[j]-u_z_n1[i];
				
				float xi_square=pow(xi_x,2)+pow(xi_y,2)+pow(xi_z,2);
				float omega=exp(-xi_square/(small_delta*small_delta));
				float xi_plus_eta=sqrt(pow((xi_x+eta_x),2)+pow((xi_y+eta_y),2)+pow((xi_z+eta_z),2));
				float e=xi_plus_eta-sqrt(xi_square);	//extension state			
				
				
				float e_d=e-(theta[i]*sqrt(xi_square)/3.0);	//deviatoric extension state
				float t=(3.0/m[i]*k_bulk_mod*theta[i]*omega*sqrt(xi_square))+(15.0*mu/m[i]*omega*e_d);
				float M_x=(xi_x+eta_x)/xi_plus_eta;
				float M_y=(xi_y+eta_y)/xi_plus_eta;
				float M_z=(xi_z+eta_z)/xi_plus_eta;
				
				f_x[i] = f_x[i]+(t*M_x*delta_V[j]);
				f_y[i] = f_y[i]+(t*M_y*delta_V[j]);
				f_z[i] = f_z[i]+(t*M_z*delta_V[j]);
				
				f_x[j] = f_x[j]-(t*M_x*delta_V[i]);
				f_y[j] = f_y[j]-(t*M_y*delta_V[i]);
				f_z[j] = f_z[j]-(t*M_z*delta_V[i]);
			}
			
		}
		
		//Calculate displacement (serialized due to slower in GPU)
		for (size_t i = 0; i < node; ++i) {
			float u_doubledot_x_n1=(f_x[i]+b_x[i])/ro; //no need to use array
			float u_doubledot_y_n1=(f_y[i]+b_y[i])/ro;
			float u_doubledot_z_n1=(f_z[i]+b_z[i])/ro;
			
			float u_dot_x_n1=u_dot_x_nhalf[i]+(delta_t/2.0*u_doubledot_x_n1); //no need to use array
			float u_dot_y_n1=u_dot_y_nhalf[i]+(delta_t/2.0*u_doubledot_y_n1);
			float u_dot_z_n1=u_dot_z_nhalf[i]+(delta_t/2.0*u_doubledot_z_n1);
			
			//Re-initialization
			
			u_x_n0[i]=u_x_n1[i];
			u_y_n0[i]=u_y_n1[i];
			u_z_n0[i]=u_z_n1[i];
			u_dot_x_n0[i]=u_dot_x_n1;
			u_dot_y_n0[i]=u_dot_y_n1;
			u_dot_z_n0[i]=u_dot_z_n1;
			u_doubledot_x_n0[i]=u_doubledot_x_n1; 
			u_doubledot_y_n0[i]=u_doubledot_y_n1;
			u_doubledot_z_n0[i]=u_doubledot_z_n1;
			
		}
		
		file_17 <<t_step<<"   "<<u_x_n1[(node/2)]<<"   "<<u_x_n1[node-1]<<endl; //disp at end of rope
		
	} //end of time integration
	file_17.close();
	
	hipDeviceSynchronize(); //CPU timer synchronization: synchronize CPU thread with GPU
	system_clock::time_point stop_parallel = system_clock::now();
	std::chrono::duration<float, std::milli> duration_parallel = stop_parallel - start_parallel;
	cout << "Parallel peridynamics = "<<duration_parallel.count()<<" millisecond"<<endl;
	
	
	for (size_t i = 0; i < node; ++i) {
		u_n1[i]=sqrt(pow(u_x_n1[i],2.0)+pow(u_y_n1[i],2.0)+pow(u_z_n1[i],2.0));
		x_plus_ux[i]=x[i]+u_x_n1[i];
		y_plus_uy[i]=y[i]+u_y_n1[i];
		z_plus_uz[i]=z[i]+u_z_n1[i];
	}
	
	
	float u_n1_sum=0.0;
	for (size_t i = 0; i < node; ++i) {
		u_n1_sum += u_n1[i];
	}
	cout<<"u_n1_sum at the end of time step = "<< u_n1_sum<<endl;
	
	
	ofstream file_18;
	file_18.open ("pos_vs_disp.txt");
	for (size_t i = 0; i < node; ++i) {
		file_18 <<x[i]<<"   "<<u_x_n1[i]<<"   "<<u_y_n1[i]<<"   "<<u_z_n1[i]<<"   "<<u_n1[i];
		if(i < (node - 1)) {
			file_18 <<endl;
		}
		
	}
	file_18.close();
	
	hipFree(buffer_neighbor_list);
	hipFree(buffer_neighbor_list_pointer);
	hipFree(buffer_delta_V); hipFree(buffer_theta); hipFree(buffer_m);
	hipFree(buffer_x); hipFree(buffer_y); hipFree(buffer_z);
	hipFree(buffer_u_x_n0); hipFree(buffer_u_y_n0); hipFree(buffer_u_z_n0);
	hipFree(buffer_u_x_n1); hipFree(buffer_u_y_n1); hipFree(buffer_u_z_n1);
	hipFree(buffer_u_dot_x_n0); hipFree(buffer_u_dot_y_n0); hipFree(buffer_u_dot_z_n0);
	hipFree(buffer_u_dot_x_nhalf); hipFree(buffer_u_dot_y_nhalf); hipFree(buffer_u_dot_z_nhalf);
	hipFree(buffer_u_doubledot_x_n0); hipFree(buffer_u_doubledot_y_n0); hipFree(buffer_u_doubledot_z_n0);
	hipFree(buffer_iter_neighbor_list); hipFree(buffer_node); 
	hipFree(buffer_small_delta); hipFree(buffer_delta_t);
	hipFree(buffer_delta_x); hipFree(buffer_delta_y);  hipFree(buffer_delta_z);
	hipFree(buffer_ndivx); hipFree(buffer_ndivy);  hipFree(buffer_ndivz);
	
	
	delete [] neighbor_list_pointer;
	delete [] neighbor_list;
	delete [] x_plus_ux;	delete [] y_plus_uy; 	delete [] z_plus_uz;
	delete [] u_n1;
	
	hipHostFree(x); hipHostFree(y); hipHostFree(z);
	hipHostFree(delta_V);
	hipHostFree(m); hipHostFree(theta);
	hipHostFree(f_x); hipHostFree(f_y); hipHostFree(f_z);
	hipHostFree(b_x); hipHostFree(b_y); hipHostFree(b_z);
	hipHostFree(u_x_n0); hipHostFree(u_y_n0); hipHostFree(u_z_n0);
	hipHostFree(u_x_n1); hipHostFree(u_y_n1); hipHostFree(u_z_n1);
	hipHostFree(u_dot_x_n0); hipHostFree(u_dot_y_n0); hipHostFree(u_dot_z_n0);
	hipHostFree(u_dot_x_nhalf); hipHostFree(u_dot_y_nhalf); hipHostFree(u_dot_z_nhalf);
	hipHostFree(u_doubledot_x_n0); hipHostFree(u_doubledot_y_n0); hipHostFree(u_doubledot_z_n0);
	
	printf("End of program!");
	
	
}